/*
 * compile with:  nvcc -arch=sm_20 -o bigMatrix bigMatrix.cu
 * run with ./bigMatrix file1, file2
 */


#include <hip/hip_runtime.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <omp.h>
#include <string.h>

//computes ab
//now, b is transposed so the task is now to multiply the rows of b with the rows of a. note: b_r and b_c are rows and columns of original b matrix
//get one processor for each item in result matrix
__global__ void mult(int *a, int *b, int *ret, int a_r, int a_c, int b_r, int b_c){

	int t = blockDim.x * blockIdx.x + threadIdx.x;
	
	int x = t / a_r; // row in a
	int y = t % b_c; //row in b_t

	int retSize = a_c * b_r;
	int sum = 0;

	int counter = 0;
	int *l = (int*)malloc(sizeof(int) * a_c);
	int *r = (int*)malloc(sizeof(int) * b_r);

	while( t < retSize ){
		for(counter = 0; counter < a_c; counter++){
			l[counter] = a[ (a_c) * x + counter];
		}

		for(counter = 0; counter < b_r; counter++){
			r[counter] = b[ (b_r) * y + counter];
		}

		if(b_r != a_c){ 
			ret[t] = -666;
		}
		else {
			for(counter = 0; counter < b_r; counter++){
				sum += l[counter] * r[counter];
			}
			ret[t] = sum;
		}

		t += gridDim.x * blockDim.x; 
		x = t / a_r; // row in a
		y = t % b_c; //row in b_t

		sum = 0;
		
	}


	free(l);
	free(r);

}

int main(int argc, char **argv){

	double start = omp_get_wtime();

	FILE *left = fopen(argv[1], "r");
	assert(left != NULL);
	FILE *right = fopen(argv[2], "r");
	assert(right != NULL);

	int l_c = 0, l_r = 0, r_c = 0, r_r = 0;

	//read rows and columns of left
	fscanf(left, "%d", &l_r);
	fscanf(left, "%d", &l_c);

	//read rows and columns of right
	fscanf(right, "%d", &r_r);
	fscanf(right, "%d", &r_c);


	int lNum = l_r * l_c;
	int rNum = r_r * r_c;
	int lrNum = l_c * r_r;

	int *lMat = (int*)malloc(sizeof(int) * lNum);
	int *rMat = (int*)malloc(sizeof(int) * rNum);
	int *lrMat = (int*)malloc(sizeof(int) * lrNum);

	int *rTranspose = (int*)malloc(sizeof(int) * rNum);
	int tranRow = 0;
	int tranCol = 0;

	//read in left matrix (argv[1])
	int counter = 0, inner = 0;
	while(counter < lNum){ fscanf(left, "%d", &lMat[counter]);  counter++;}

	//read in right matrix (argv[2])
	counter = 0;
	while(counter < rNum){ 
		fscanf(right, "%d", &rMat[counter]);  
		rTranspose[ (r_r * tranRow) + tranCol] = rMat[counter]; //transpose matrix
		counter++;

		tranRow++;
		if(tranRow == r_c){
			tranRow = 0;
			tranCol++;
		}
	}


//have to manually set heap size http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#dynamic-global-memory-allocation-and-operations
/*
	to multiply a mXn and a oXp matrix, need the following memory: mXn + oXp + mXp + (m + p)(mXp)
*/
	int *a, *b, *ret;
	assert( hipMalloc(&a, sizeof(int) * lNum) != hipErrorOutOfMemory);
	assert(hipMalloc(&b, sizeof(int) * rNum) != hipErrorOutOfMemory);
	assert(hipMalloc(&ret, sizeof(int) * lrNum) != hipErrorOutOfMemory);

	assert( hipMemcpy(a, lMat, sizeof(int) * lNum, hipMemcpyHostToDevice) == hipSuccess);
	assert( hipMemcpy(b, rTranspose, sizeof(int) * rNum, hipMemcpyHostToDevice) == hipSuccess);

	//printf(cudaGetErrorString(cudaMemcpy(b, rTranspose, sizeof(int) * rNum, cudaMemcpyHostToDevice) ));// == cudaSuccess);

	mult<<<12, 128>>>(a, b, ret, l_r, l_c, r_r, r_c); //switch the rows and columns for the transposed matrix

printf(hipGetErrorString(hipMemcpy(lrMat, ret, sizeof(int) * lrNum, hipMemcpyDeviceToHost) ));
	assert(hipMemcpy(lrMat, ret, sizeof(int) * lrNum, hipMemcpyDeviceToHost) == hipSuccess);

	//print result matrix
	FILE *out = fopen("cudamat", "w");
	for(counter = 0; counter < l_r; counter++){
		for(inner = 0; inner < r_c; inner++){
			fprintf(out, "%d ", lrMat[ (counter * r_c) + inner]);
		}
		fprintf(out, "\n");
	}

	fclose(out);

	hipFree(a);
	hipFree(b);
	hipFree(ret);

	free(lMat);
	free(rMat);
	free(lrMat);
	
	free(rTranspose);

	fclose(left);
	fclose(right);

	double end = omp_get_wtime();
	printf("%lf\n", end - start);

	return 0;
}
